﻿/*
**
** Parallelized implementation of Unoptimized Algorithm to generate the Mandelbrot Set.
** Written and tested on the CUDA 12.4 Toolkit
**
** Author: Tasman Grinnell
**
**/


#include "hip/hip_runtime.h"


#include <stdio.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <chrono>

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

constexpr int X = 1920;
constexpr int Y = 1080;

__device__ int xParam = X;
__device__ int yParam = Y;

struct Point {
    int x;
    int y;
    int iteration;
    int sizeX;
    int sizeY;
};

hipError_t mandelBrotCalc(struct Point* pointArray, unsigned long size);

__global__ void computeSet(struct Point* returnPointArr) {   

    // Calculates Strides by finding the area a block is supposed to work on and add an offset based on the thread ID.
    //                              BEGINNING OF BLOCK                 THREAD OFFSET
    int block_xStart    = xParam * ((double) blockIdx.x / gridDim.x) + xParam * threadIdx.x / blockDim.x / gridDim.x;
    int block_xEnd      = xParam * ((double) blockIdx.x / gridDim.x) + xParam * (threadIdx.x + 1) / blockDim.x / gridDim.x;

    int block_yStart    = yParam * ((double) blockIdx.y / gridDim.y) + yParam * threadIdx.y / blockDim.y / gridDim.y;
    int block_yEnd      = yParam * ((double) blockIdx.y / gridDim.y) + yParam * (threadIdx.y + 1) / blockDim.y / gridDim.y;
    
    // Unoptimized escape algorithm
    for (int i = block_xStart; i < block_xEnd; i++)
    {
        for (int j = block_yStart; j < block_yEnd; j++)
        {

            float x0 = i / (double) xParam * 2.47 - 2;
            float y0 = j / (double) yParam * 2.24 - 1.12;
            float x = 0.0;
            float y = 0.0;

            int iteration = 0;
            int max_iteration = 1000;

            while (x * x + y * y <= (2 * 2) && iteration < max_iteration)
            {
                float xtemp = x * x - y * y + x0;
                y = 2 * x * y + y0;
                x = xtemp;

                iteration++;
            }

            struct Point newPoint = { i, j, iteration, 0, 0 };

            returnPointArr[i * yParam + j] = newPoint;

        }

    }
}

/*
*
* Unoptimized escape for calculating the Mandelbrot Set
* 
*/
int main()
{
    std::cout.setf(std::ios_base::unitbuf);

    struct Point *pointArray = new struct Point[X * Y];

    std::cout << "Beginning Calculation" << std::endl;
    
    auto tStart = std::chrono::high_resolution_clock::now();
    // Add vectors in parallel.
    hipError_t cudaStatus = mandelBrotCalc(pointArray, (unsigned long) X * Y );
    auto tEnd = std::chrono::high_resolution_clock::now() ;

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mandelBrotCalc failed!");
        return 1;
    } 

    std::cout << "Ending Calculation successfully, Beginning file output" << std::endl;
    
    // Output Stream
    // TODO: Modify for actual use
    std::ofstream file("MandelSetOut_Parallel.csv");

    file << "X,Y,Iteration,sizeX,sizeY" << std::endl;

    for (int i = 0; i < X * Y; i++) {
        if (i == 0) {
            file << pointArray[i].x << "," << pointArray[i].y << "," << pointArray[i].iteration << ","
                << X << "," << Y
                << std::endl;
        }

        file << pointArray[i].x << "," << pointArray[i].y << "," << pointArray[i].iteration << ",0,0" << std::endl;
    }

    // Output results
    std::cout << "Ending file output" << std::endl;

    // cudaDeviceReset must be called before exiting 
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to calculate mandelBrot set in parallel.
hipError_t mandelBrotCalc (struct Point* pointArray, unsigned long size)
{
    struct Point *dev_points = 0;

    dim3 nthreads(16, 9);
    dim3 nblocks(6, 6);

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for output vector
    cudaStatus = hipMalloc((void**)&dev_points, X * Y * sizeof(struct Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Call kernel
    computeSet CUDA_KERNEL (nblocks, nthreads)  (dev_points);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "computeSet launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeSet!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pointArray, dev_points, X * Y * sizeof(struct Point), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy points failed!");
        goto Error;
    }

Error:
    hipFree(dev_points);
    
    return cudaStatus;
}
