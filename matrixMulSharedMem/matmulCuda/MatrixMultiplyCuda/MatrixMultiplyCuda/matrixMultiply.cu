﻿
#include "hip/hip_runtime.h"


#include <cstdlib>
#include <stdio.h>
#include <string>
#include <iostream>

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

constexpr int rows = 3;
__device__ constexpr int matRows = rows;

// Kernel wrapper class
hipError_t kernelWrapper(int outputMatrix[rows][rows]);
// Matrix Multiply kernel
// int matA[matRows][matRows], int matB[matRows][matRows],
__global__ void matMul(int matrixA[matRows][matRows], int matrixB[matRows][matRows], int outputMat[matRows][matRows]);

// Helper functions to set up and print matrices
void randMatVals(int matrix[rows][rows]);
void printMat(int mat[rows][rows], std::string string);
void clearMat(int matrix[rows][rows]);

int main()
{
    int outputMatrix[rows][rows];

    // Initializes Matrices with random values

    // Sets all values in outputMatrix to 0
    clearMat(outputMatrix);

    // Add vectors in parallel.
    hipError_t cudaStatus = kernelWrapper(outputMatrix);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // Free Cuda
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t kernelWrapper(int outputMatrix[rows][rows])
{
    dim3 blockDims(matRows, matRows);
    int dev_matA[matRows][matRows];
    int dev_matB[matRows][matRows];

    randMatVals(dev_matA);
    randMatVals(dev_matB);

    printMat(dev_matA, "Matrix A");
    printMat(dev_matB, "Matrix B");
    

    int dev_outMat[rows][rows];
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Set shared Memory for matricesA, B
    cudaStatus = hipMallocManaged((void**)&dev_matA, sizeof(dev_matA));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocManaged((void**)&dev_matB, sizeof(dev_matB));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Allocate buffer for output Matrix
    cudaStatus = hipMalloc((void**)&dev_outMat, sizeof(outputMatrix));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy cleared output matrix into the dev_outMat 
    cudaStatus = hipMemcpy(dev_outMat, outputMatrix, sizeof(outputMatrix), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    matMul CUDA_KERNEL(1, blockDims) (dev_matA, dev_matB, dev_outMat);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(outputMatrix, dev_outMat, rows * rows * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    printMat(dev_outMat, "Output Matrix");

Error:
    hipFree(dev_matA);
    hipFree(dev_matB);
    hipFree(dev_outMat);
    
    return cudaStatus;
}

__global__ void matMul(int matrixA[matRows][matRows], int matrixB[matRows][matRows], int outputMat[matRows][matRows]) {

    for (int i = 0; i < matRows; i++) {
        outputMat[threadIdx.x][threadIdx.y] += matrixA[threadIdx.x][i] * matrixB[i][threadIdx.y];
    }

}

void randMatVals(int matrix[rows][rows]) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < rows; j++) {
            // generates random values for a matrix between 0-200
            matrix[i][j] = rand() % 10;
        }
    }
}

void printMat(int mat[rows][rows], std::string string) {

    std::cout << string << std::endl;

    for (int i = 0; i < rows; i++) {
        std::cout << "[ ";

        for (int j = 0; j < rows; j++) {
            std::cout << mat[i][j];
            if (j != rows - 1) {
                std::cout << "\t";
            }
        }

        std::cout << " ]" << std::endl;
    }

}

void clearMat(int matrix[rows][rows]) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < rows; j++) {
            matrix[i][j] = 0;
        }
    }
}