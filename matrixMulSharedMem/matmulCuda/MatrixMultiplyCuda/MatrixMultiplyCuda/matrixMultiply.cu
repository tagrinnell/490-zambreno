﻿#include "hip/hip_runtime.h"


#include <cstdlib>
#include <stdio.h>
#include <string>
#include <iostream>

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

constexpr int rows = 3;
__device__ constexpr int matRows = rows;

__managed__ int matrixA[rows][rows];
__managed__ int matrixB[rows][rows];

// Kernel wrapper class
hipError_t kernelWrapper();
// Matrix Multiply kernel
// int matA[matRows][matRows], int matB[matRows][matRows],
__global__ void matMul(int outputMat[matRows][matRows]);

// Helper functions to set up and print matrices
void randMatVals(int matrix[rows][rows]);
void printMat(int mat[rows][rows], std::string string);
void clearMat(int matrix[rows][rows]);

int main()
{
    // Add vectors in parallel.
    hipError_t cudaStatus = kernelWrapper();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // Free Cuda
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t kernelWrapper()
{
    dim3 blockDims(rows, rows);
    int dev_outMat[rows][rows];
    int matrix[rows][rows];
    clearMat(matrix);

    // Set up device __managed__ matrices
    randMatVals(matrixA);
    randMatVals(matrixB);

    printMat(matrixA, "__managed__ Matrix A");
    printMat(matrixB, "__managed__ Matrix B");
    printMat(matrix, "Output Matrix");

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate buffer for output Matrix
    cudaStatus = hipMalloc((void**)&dev_outMat, rows * rows * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //clearMat(dev_outMat);

    // Copy cleared output matrix into the dev_outMat 
    /*cudaStatus = cudaMemcpy(*dev_outMat, *matrix, rows * rows * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "To device cudaMemcpy failed!");
        goto Error;
    }*/

    // Launch a kernel on the GPU with one thread for each element.
    matMul CUDA_KERNEL(1, blockDims) (dev_outMat);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(matrix, dev_outMat, rows * rows * sizeof(int), hipMemcpyDeviceToHost);
    /*if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy to host failed!");
        goto Error;
    }*/

    printMat(matrix, "Output Matrix");

Error:
    hipFree(dev_outMat);
    
    return cudaStatus;
}

__global__ void matMul(int outputMat[matRows][matRows]) {
    outputMat[threadIdx.x][threadIdx.y] = 0;

    for (int i = 0; i < matRows; i++) {
        outputMat[threadIdx.x][threadIdx.y] += matrixA[threadIdx.x][i] * matrixB[i][threadIdx.y];
    }

}

void randMatVals(int matrix[rows][rows]) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < rows; j++) {
            // generates random values for a matrix between 0-200
            matrix[i][j] = rand() % 10;
        }
    }
}

void printMat(int mat[rows][rows], std::string string) {

    std::cout << string << std::endl;

    for (int i = 0; i < rows; i++) {
        std::cout << "[ ";

        for (int j = 0; j < rows; j++) {
            std::cout << mat[i][j];
            if (j != rows - 1) {
                std::cout << "\t";
            }
        }

        std::cout << " ]" << std::endl;
    }

}

void clearMat(int matrix[rows][rows]) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < rows; j++) {
            matrix[i][j] = 0;
        }
    }
}